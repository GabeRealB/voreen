#include "lib.h"
#include "network.h"
#include <hip/hip_runtime.h>
#include "../../datastructures/vesselgraph.h"

#include <iostream>

stim::network<float> GT;			// ground truth network
stim::network<float> T;			// test network

int device = -1;

void setdevice(){
	int count;
	hipGetDeviceCount(&count);					// numbers of device that are available
	if(count > 0) {
		device = 1;
	} else {
		std::cout<<"No cuda device available."<<std::endl;
		device = -1;
	}
}

float netmets_compare_networks(const voreen::VesselGraph& g1, const voreen::VesselGraph& g2) {
    if(device == -1) {
	setdevice();
    }
    // find appropriate radius
    float radiusSum = 0.0;
    for(const auto& edge : g1.getEdges()) {
        radiusSum += edge.getAvgRadiusAvg();
    }
    for(const auto& edge : g2.getEdges()) {
        radiusSum += edge.getAvgRadiusAvg();
    }
    float globalAvgRadius = radiusSum / (g1.getEdges().size() + g2.getEdges().size());
    float sigma = globalAvgRadius;

    GT = GT.compare(T, sigma, device);				// compare the ground truth to the test case - store errors in GT
T = T.compare(GT, sigma, device);				// compare the test case to the ground truth - store errors in T

    //calculate the metrics
    float FPR = GT.average();						// calculate the metrics
    float FNR = T.average();

    return FPR*FNR;
}
