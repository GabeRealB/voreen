#include "lib.h"
#include "network.h"
#include <hip/hip_runtime.h>
#include "../../datastructures/vesselgraph.h"

#include <iostream>

int netmetsCudaDevice = -1;

void setdevice(){
    int count;
    hipGetDeviceCount(&count);                    // numbers of device that are available
    if(count > 0) {
        netmetsCudaDevice = 1;
    } else {
        std::cout<<"No cuda device available."<<std::endl;
        netmetsCudaDevice = -1;
    }
}

typedef stim::network<float> Network;

Network networkFromVesselGraph(const voreen::VesselGraph& g) {
    Network output;

    for(const auto& node : g.getNodes()) {
        if(node.getDegree() == 0) {
            // Freestanding nodes are not handled well by netmets
            continue;
        }

        stim::vec3<float> pos(node.pos_.x, node.pos_.y, node.pos_.z);
        Network::vertex new_vertex(pos);

        for(const auto& edge : g.getEdges()) {
            if(edge.getNodeID1() == node.getID()) {
                new_vertex.e[0].push_back(edge.getID().raw());
            }
            if(edge.getNodeID2() == node.getID()) {
                new_vertex.e[1].push_back(edge.getID().raw());
            }
        }
        output.V.push_back(new_vertex);
    }

    for(const auto& edge : g.getEdges()) {
        const auto& voxels = edge.getVoxels();
        size_t centerline_length = voxels.size() + 2;
        stim::centerline<float> c3(centerline_length);
        std::vector<float> radius(c3.size());

        c3[0] = stim::vec3<float>(edge.getNode1().pos_.x, edge.getNode1().pos_.y, edge.getNode1().pos_.z);
        radius[0] = edge.getNode1().getRadius(); //TODO
        radius[0] = 0.0;
        int i = 1;
        for(const auto& voxel : voxels) {
            stim::vec3<float> pos(voxel.pos_.x, voxel.pos_.y, voxel.pos_.z);
            c3[i] = pos;
            if(voxel.hasValidData()) {
                radius[i] = voxel.avgDistToSurface_;
            } else {
                radius[i] = 0.0; //TODO maybe think of something else
            }
            ++i;
        }
        c3[centerline_length - 1] = stim::vec3<float>(edge.getNode2().pos_.x, edge.getNode2().pos_.y, edge.getNode2().pos_.z);
        radius[centerline_length - 1] = edge.getNode2().getRadius();
        radius[centerline_length - 1] = 0.0;

        c3.update();

        stim::cylinder<float> C3(c3);
        C3.copy_r(radius);

        Network::edge new_edge(C3);
        new_edge.v[0] = edge.getNodeID1().raw();
        new_edge.v[1] = edge.getNodeID2().raw();

        output.E.push_back(new_edge);
    }

    return output;
}

NetmetsResult netmets_compare_networks(const voreen::VesselGraph& groundtruthNetwork, const voreen::VesselGraph& testNetwork) {
    if(netmetsCudaDevice == -1) {
        setdevice();
    }
    // find appropriate radius
    float radiusSum = 0.0;
    size_t numConsideredVoxels = 0;
    auto processGraph = [&] (const voreen::VesselGraph& g) {
        for(const auto& edge : g.getEdges()) {
            for(const auto& voxel : edge.getVoxels()) {
                if(voxel.hasValidData()) {
                    radiusSum += voxel.avgDistToSurface_;
                    ++numConsideredVoxels;
                }
            }
        }
    };
    processGraph(groundtruthNetwork);
    float globalAvgRadius = radiusSum / numConsideredVoxels;
    float sigma = globalAvgRadius * 10.0 /*Found experimentally to yield good results in netmets application */;
    std::cout << "Using sigma: " << sigma << std::endl;

    auto gtnm = networkFromVesselGraph(groundtruthNetwork);
    auto tnm = networkFromVesselGraph(testNetwork);

    // For debugging purposes:
    //gtnm.saveNwt("/home/dominik/g1.nwt");
    //tnm.saveNwt("/home/dominik/g2.nwt");

    // TODO check that overwriting is fine
    gtnm = gtnm.compare(tnm, sigma, netmetsCudaDevice);                // compare the ground truth to the test case - store errors in GT
    tnm = tnm.compare(gtnm, sigma, netmetsCudaDevice);                // compare the test case to the ground truth - store errors in T

    NetmetsResult result;
    //calculate the metrics TODO: Make sure that fnr/fpr are in the right order here
    result.fnr = gtnm.average();
    result.fpr = tnm.average();

    return result;
}
